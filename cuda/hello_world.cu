#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void print_kernel()
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    hipDeviceReset();
    print_kernel<<<10,10>>>();
    hipError_t err = hipDeviceSynchronize();
    printf("Error %s\n", hipGetErrorString(err));
    return 0;
}
